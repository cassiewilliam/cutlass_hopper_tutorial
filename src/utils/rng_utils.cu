#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "utils/rng_utils.h"

#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <iostream>

#define _CG_ABI_EXPERIMENTAL
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>

namespace turbomind {

namespace cg = cooperative_groups;

__global__ void hiprand_init(hiprandState* state)
{
    auto tid = cg::this_grid().thread_rank();
    hiprand_init(0xe4c45822e90461ddULL, tid, 0, state + tid);
}

template<typename T>
__global__ void hiprand_uniform(hiprandState* state, size_t count, T* result, float scale, float shift)
{
    auto grid = cg::this_grid();
    for (auto i = grid.thread_rank(); i < count; i += grid.size()) {
        float tmp = hiprand_uniform(state + grid.thread_rank());
        result[i] = T(scale * tmp + shift);
    }
}

template<typename T>
__global__ void hiprand_normal(hiprandState* state, size_t count, T* result, float scale, float shift)
{
    auto grid = cg::this_grid();
    for (auto i = grid.thread_rank(); i < count; i += grid.size()) {
        float tmp = hiprand_normal(state + grid.thread_rank());
        result[i] = T(scale * tmp + shift);
    }
}

__global__ void curand_bytes(hiprandState* state, size_t count, uint* result)
{
    auto grid = cg::this_grid();
    for (auto i = grid.thread_rank(); i < count; i += grid.size()) {
        result[i] = hiprand(state + grid.thread_rank());
    }
}

struct RNG::Impl {

    hiprandState* states{};

    Impl()
    {
        hipMalloc(&states, sizeof(hiprandState) * 64 * 64);
        hiprand_init<<<64, 64>>>(states);
    }

    ~Impl()
    {
        hipFree(states);
    }

    void GenerateUInt(uint* out, size_t count)
    {
        curand_bytes<<<64, 64, 0, stream_>>>(states, count, out);
    }

    template<typename T>
    void GenerateUniform(T* out, size_t count, float scale, float shift)
    {
        hiprand_uniform<<<64, 64, 0, stream_>>>(states, count, out, scale, shift);
    }

    template<typename T>
    void GenerateNormal(T* out, size_t count, float scale, float shift)
    {
        hiprand_normal<<<64, 64, 0, stream_>>>(states, count, out, scale, shift);
    }

    hipStream_t stream_{};
};

RNG::RNG(): impl_(std::make_unique<Impl>()) {}

RNG::~RNG() = default;

void RNG::GenerateUInt(uint* out, size_t count)
{
    impl_->GenerateUInt(out, count);
}

template<typename T>
void RNG::GenerateUniform(T* out, size_t count, float scale, float shift)
{
    impl_->GenerateUniform(out, count, scale, shift);
}

template<typename T>
void RNG::GenerateNormal(T* out, size_t count, float scale, float shift)
{
    impl_->GenerateNormal(out, count, scale, shift);
}

hipStream_t RNG::stream() const
{
    return impl_->stream_;
}

void RNG::set_stream(hipStream_t stream)
{
    impl_->stream_ = stream;
}

template void RNG::GenerateUniform(half* out, size_t count, float scale, float shift);
template void RNG::GenerateUniform(float* out, size_t count, float scale, float shift);
#if ENABLE_BF16
template void RNG::GenerateUniform(hip_bfloat16* out, size_t count, float scale, float shift);
#endif

template void RNG::GenerateNormal(half* out, size_t count, float scale, float shift);
template void RNG::GenerateNormal(float* out, size_t count, float scale, float shift);
#if ENABLE_BF16
template void RNG::GenerateNormal(hip_bfloat16* out, size_t count, float scale, float shift);
#endif

}  // namespace turbomind
