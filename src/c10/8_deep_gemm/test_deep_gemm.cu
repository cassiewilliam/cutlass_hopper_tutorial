#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "jit_utils.cuh"
#include "deep_gemm.h"

float max_absolute_error(const thrust::host_vector<__hip_bfloat16>& ref,
                         const thrust::host_vector<__hip_bfloat16>& result)
{
    if (ref.size() != result.size())
    {
        std::cerr << "Error: Tensor size mismatch! (" << ref.size() << " vs " << result.size() << ")\n";
        return -1.0f;
    }

    return thrust::transform_reduce(ref.begin(), ref.end(), result.begin(),
            [](const __hip_bfloat16& a, const __hip_bfloat16& b) {
                return fabs(__bfloat16_to_float(a) - __bfloat16_to_float(b));
            }, 0.0f, thrust::maximum<float>());
}

int main()
{
    int m = 256;
    int n = 5120;
    int k = 5120;

    auto gemm_runner = std::make_shared<c108::DeepGemmRunner>();

    deep_gemm::GemmType gemm_type = deep_gemm::GemmType::Normal;
    gemm_runner->tunning(m, n, k, gemm_type);

    thrust::host_vector<float> host_A(m * k);
    thrust::host_vector<float> host_B(n * k);
    thrust::host_vector<float> host_C(m * n);

    // Initialize the tensors
    c108::random_initialize(host_A.data(), m * k, -128.0f, 128.0f);
    c108::random_initialize(host_B.data(), n * k, -128.0f, 128.0f);
    for (int j = 0; j < m*n; ++j)
        host_C[j] = float(0);

    thrust::device_vector<float> device_A = host_A;
    thrust::device_vector<float> device_B = host_B;
    thrust::device_vector<float> device_C = host_C;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    c108::matmul_cublas(device_A.data(),
                        device_B.data(),
                        device_C.data(),
                        m, k, n, handle);
    hipblasDestroy(handle);

    thrust::device_vector<__hip_fp8_e4m3_fnuz> lhs_activation_a_x_fp8(m * k);
    thrust::device_vector<float> lhs_activation_a_x_scales(m * ceil_div(k, 128) * 128);
    c108::per_token_cast_to_fp8(device_A.data(),
                                lhs_activation_a_x_scales.data(),
                                lhs_activation_a_x_fp8.data(),
                                m,
                                k,
                                stream);

    thrust::device_vector<__hip_fp8_e4m3_fnuz> rhs_weight_b_y_fp8(n * k);
    thrust::device_vector<float> rhs_weight_b_y_scales(ceil_div(n, 128) * 128 * ceil_div(k, 128) * 128);
    c108::per_block_cast_to_fp8(device_B.data(),
                                rhs_weight_b_y_scales.data(),
                                rhs_weight_b_y_fp8.data(),
                                n,
                                k,
                                stream);

    thrust::device_vector<__hip_bfloat16> result(m * n);

    gemm_runner->gemm(m, n, k,
                      (void *)(lhs_activation_a_x_fp8.data()),
                      (void *)(lhs_activation_a_x_scales.data()),
                      (void *)(rhs_weight_b_y_fp8.data()),
                      (void *)(rhs_weight_b_y_scales.data()),
                      (void *)(result.data()));

    thrust::host_vector<__hip_bfloat16> host_result_ref = device_C;
    thrust::host_vector<__hip_bfloat16> host_result = result;

    std::cout << max_absolute_error(host_result_ref, host_result);
}

