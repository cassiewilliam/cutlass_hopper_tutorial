#pragma once

#include <hip/hip_runtime.h>
#include <iostream>

#include <cutlass/cutlass.h>

#include <cute/tensor.hpp>
#include <cutlass/tensor_ref.h>
#include <cutlass/epilogue/collective/default_epilogue.hpp>
#include <cutlass/epilogue/thread/linear_combination.h>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/epilogue/collective/collective_builder.hpp>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/gemm/kernel/tile_scheduler_params.h>

#include <cutlass/util/command_line.h>
#include <cutlass/util/distribution.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/packed_stride.hpp>
#include <cutlass/util/tensor_view_io.h>
#include <cutlass/util/reference/device/gemm.h>
#include <cutlass/util/reference/device/tensor_compare.h>
#include <cutlass/util/reference/device/tensor_fill.h>

namespace c1000
{
/**
 * Panic wrapper for unwinding CUTLASS errors
 */
#define CUTLASS_CHECK(status)                                                                    \
  {                                                                                              \
    cutlass::Status error = status;                                                              \
    if (error != cutlass::Status::kSuccess) {                                                    \
      std::cerr << "Got cutlass error: " << cutlassGetStatusString(error) << " at: " << __LINE__ \
                << std::endl;                                                                    \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }


/**
 * Panic wrapper for unwinding CUDA runtime errors
 */
#define CUDA_CHECK(status)                                              \
  {                                                                     \
    hipError_t error = status;                                         \
    if (error != hipSuccess) {                                         \
      std::cerr << "Got bad cuda status: " << hipGetErrorString(error) \
                << " at line: " << __LINE__ << std::endl;               \
      exit(EXIT_FAILURE);                                               \
    }                                                                   \
  }


/**
 * GPU timer for recording the elapsed time across kernel(s) launched in GPU stream
 */
struct GpuTimer
{
    hipStream_t _stream_id;
    hipEvent_t _start;
    hipEvent_t _stop;

    /// Constructor
    GpuTimer() : _stream_id(0)
    {
        CUDA_CHECK(hipEventCreate(&_start));
        CUDA_CHECK(hipEventCreate(&_stop));
    }

    /// Destructor
    ~GpuTimer()
    {
        CUDA_CHECK(hipEventDestroy(_start));
        CUDA_CHECK(hipEventDestroy(_stop));
    }

    /// Start the timer for a given stream (defaults to the default stream)
    void start(hipStream_t stream_id = 0)
    {
        _stream_id = stream_id;
        CUDA_CHECK(hipEventRecord(_start, _stream_id));
    }

    /// Stop the timer
    void stop()
    {
        CUDA_CHECK(hipEventRecord(_stop, _stream_id));
    }

    /// Return the elapsed time (in milliseconds)
    float elapsed_millis()
    {
        float elapsed = 0.0;
        CUDA_CHECK(hipEventSynchronize(_stop));
        CUDA_CHECK(hipEventElapsedTime(&elapsed, _start, _stop));
        return elapsed;
    }
};



}